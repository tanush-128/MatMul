#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <random>
#include ""
#include <stdio.h>
#include <stdlib.h>

void fillMatrix(int* matrix, int N)
{
    for (int i = 0; i < N * N; i++)
    {
        matrix[i] = rand() % 100; // Fill with random numbers between 0 and 99
    }
}

void printMatrix(int* matrix, int N)
{
    for (int i = 0; i < N * N; i++)
    {
        printf("%d ", matrix[i]);
        if ((i + 1) % N == 0)
        {
            printf("\n");
        }
    }
}
__global__ void matrixMul(int* a, int* b, int* c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    if (row < N && col < N)
    {
        for (int i = 0; i < N; i++)
        {
            temp_sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = temp_sum;
    }
}

void cpuMatrixMul(int* a, int* b, int* c, int N)
{
    int i, j, k;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            int sum = 0;
            for (k = 0; k < N; k++)
            {
                sum += a[i * N + k] * b[k * N + j];
            }
            c[i * N + j] = sum;
        }
    }
}

void checkError(int* A, int* B, int* C, int N)
{
    int i, j, k;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            int sum = 0;
            for (k = 0; k < N; k++)
            {
                sum += A[i * N + k] * B[k * N + j];
            }
            if (C[i * N + j] != sum)
            {
                printf("Error at C[%d][%d] = %d\n", i, j, C[i * N + j]);
            }
        }
    }
    printf("Check complete\n");
}

int main()
{
    // Initialize matrices a, b and c on the host and device
    int N = 1024;
    int size = N * N * sizeof(int);
    int* h_a, * h_b, * h_c;
    int* d_a, * d_b, * d_c;

    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    fillMatrix(h_a, N);
    fillMatrix(h_b, N);

    /*
    printf("Matrix A:\n");
    printMatrix(h_a, N);
    printf("Matrix B:\n");
    printMatrix(h_b, N);
    */

    clock_t start1, end1;
    double cpu_time_used;
    start1 = clock();
    // Perform matrix multiplication on the CPU
    cpuMatrixMul(h_a, h_b, h_c, N);
    end1 = clock();
    cpu_time_used = ((double)(end1 - start1)) / CLOCKS_PER_SEC;
    printf("The elapsed time in cpu was %.2f ms\n", cpu_time_used * 1000);
    checkError(h_a, h_b, h_c, N);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    matrixMul << <numBlocks, threadsPerBlock >> > (d_a, d_b, d_c, N);

    // Record the stop event
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("The elapsed time in gpu was %.2f ms\n", milliseconds);

    // Copy the result back to the host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Check for errors
    checkError(h_a, h_b, h_c, N);

    /*
    printf("Matrix C:\n");
    printMatrix(h_c, N);
    */

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
